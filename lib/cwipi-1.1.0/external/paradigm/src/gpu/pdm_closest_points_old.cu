
#include <hip/hip_runtime.h>
// /*
//   This file is part of the ParaDiGM library.

//   Copyright (C) 2019       ONERA

//   This library is free software; you can redistribute it and/or
//   modify it under the terms of the GNU Lesser General Public
//   License as published by the Free Software Foundation; either
//   version 2.1 of the License, or (at your option) any later version.

//   This library is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
//   Lesser General Public License for more details.

//   You should have received a copy of the GNU Lesser General Public
//   License along with this library; if not, write to the Free Software
//   Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA  02110-1301  USA
// */
// /*----------------------------------------------------------------------------
//  * System headers
//  *----------------------------------------------------------------------------*/

// #include <assert.h>
// #include <stdio.h>
// #include <math.h>
// #include <string.h>
// #include <stdlib.h>

// /*----------------------------------------------------------------------------
//  * Local headers
//  *----------------------------------------------------------------------------*/

// #include "pdm_cuda_error.cuh"
// #include "pdm_cuda.cuh"
// #include "pdm_printf.h"
// #include "pdm_error.h"
// #include "pdm.h"
// #include "pdm_priv.h"
// #include "pdm_handles.h"
// #include "pdm_handles.cuh"
// #include "pdm_mpi.h"
// #include "pdm_timer.h"
// #include "pdm_timer.cuh"
// #include "pdm_closest_points.h"
// #include "pdm_closest_points.cuh"
// #include "pdm_para_octree.h"
// #include "pdm_para_octree.cuh"

// /*----------------------------------------------------------------------------*/

// #ifdef	__cplusplus
// extern "C" {
// #if 0
// } /* Fake brace to force Emacs auto-indentation back to column 0 */
// #endif
// #endif


// /*============================================================================
//  * Macro definitions
//  *============================================================================*/


// #define NTIMER 2

// /*============================================================================
//  * Type definitions
//  *============================================================================*/


// /**
//  * \enum _timer_step_t
//  *
//  */

// typedef enum {

//   BEGIN    = 0,
//   END      = 1,

// } _timer_step_t;


// /**
//  * \struct _tgt_point_cloud_t
//  * \brief  Target point cloud structure
//  *
//  */

// typedef struct {

//   int           n_part;            /*!< Number of partition */
//   int          *n_points;          /*!< Number of points of each partition */
//   double      **coords;            /*!< Point coordinates points of each partition */
//   PDM_g_num_t **gnum;              /*!< Point global numbering of each partition */
//   PDM_g_num_t **closest_src_gnum;  /*!< Global numbering of the n_closest source points
//                                      for each point of each partition  */
//   double      **closest_src_dist; /*!< Distance to the n_closest source points
//                                     for each point of each partition  */

// } _tgt_point_cloud_t;


// /**
//  * \struct _src_point_cloud_t
//  * \brief  Src point cloud structure
//  *
//  */

// typedef struct {

//   int           n_part;            /*!< Number of partition */
//   int          *n_points;          /*!< Number of points of each partition */
//   double      **coords;            /*!< Point coordinates points of each partition */
//   PDM_g_num_t **gnum;              /*!< Point global numbering of each partition */

// } _src_point_cloud_t;


// /**
//  * \struct _PDM_closest_t
//  * \brief  Closest points structure
//  *
//  */

// typedef struct {

//   PDM_MPI_Comm comm;  /*!< MPI communicator */

//   int n_closest;  /*!< Number of closest source points to find for each
//                     target point  */

//   _src_point_cloud_t *src_cloud; /*!< Source point cloud */

//   _tgt_point_cloud_t *tgt_cloud; /*!< Target point cloud */

//   PDM_timer_t *timer; /*!< Timer */

//   double times_elapsed[NTIMER]; /*!< Elapsed time */

//   double times_cpu[NTIMER];     /*!< CPU time */

//   double times_cpu_u[NTIMER];  /*!< User CPU time */

//   double times_cpu_s[NTIMER];  /*!< System CPU time */


// } _PDM_closest_t;


// /*============================================================================
//  * Global variable
//  *============================================================================*/

// static PDM_Handles_t *_closest_pts   = NULL;

// static int idebug = 0;

// /*=============================================================================
//  * Private function definitions
//  *============================================================================*/

// /**
//  *
//  * \brief Return ppart object from it identifier
//  *
//  * \param [in]   ppart_id        ppart identifier
//  *
//  */

// static _PDM_closest_t *
// _get_from_id
// (
//  int  id
//  )
// {
//   printf("before handle get\n");
//   printf("id = %d\n", id);
//   printf("s array = %d\n", _closest_pts->s_array);
//   _PDM_closest_t *closest = (_PDM_closest_t *) PDM_Handles_get (_closest_pts, id);

//   if (closest == NULL) {
//     PDM_error(__FILE__, __LINE__, 0, "PDM_closest_points error : Bad identifier\n");
//   }

//   return closest;
// }


// /*============================================================================
//  * Public function definitions
//  *============================================================================*/



// /**
//  *
//  * \brief Set the number of partitions of a point cloud
//  *
//  * \param [in]   id                Identifier
//  * \param [in]   n_part_cloud_src  Number of partitions of the source cloud
//  * \param [in]   n_part_cloud_tgt  Number of partitions of the target cloud
//  *
//  */

// /* void
// PDM_closest_points_n_part_cloud_set
// (
//  const int  id,
//  const int  n_part_cloud_src,
//  const int  n_part_cloud_tgt
//  )
// {
//   _PDM_closest_t *cls = _get_from_id (id);
//   assert(cls->src_cloud == NULL);
//   assert(cls->tgt_cloud == NULL);

//   cls->src_cloud = malloc (sizeof(_src_point_cloud_t));
//   cls->tgt_cloud = malloc (sizeof(_tgt_point_cloud_t));

//   cls->src_cloud->n_part = n_part_cloud_src;
//   cls->src_cloud->coords = malloc (sizeof(double *) * n_part_cloud_src);
//   cls->src_cloud->gnum = malloc (sizeof(int *) * n_part_cloud_src);
//   cls->src_cloud->n_points = malloc (sizeof(int) * n_part_cloud_src);

//   cls->tgt_cloud->n_part = n_part_cloud_tgt;
//   cls->tgt_cloud->coords = malloc (sizeof(double *) * n_part_cloud_tgt);
//   cls->tgt_cloud->gnum = malloc (sizeof(int *) * n_part_cloud_tgt);
//   cls->tgt_cloud->n_points = malloc (sizeof(int) * n_part_cloud_tgt);
//   cls->tgt_cloud->closest_src_gnum = NULL;
//   cls->tgt_cloud->closest_src_dist = NULL;
// } */


// /**
//  *
//  * \brief Look for closest points
//  *
//  * \param [in]   id  Identifier
//  *
//  */

// void
// PDM_closest_points_compute_GPU
// (
//  const int id,
//  PDM_Handles_t *var
//  )
// {
//   _closest_pts = var;
//   printf("s array = %d\n", _closest_pts->s_array);
//   _PDM_closest_t *cls = NULL;
//   int *octree_id = NULL;

//   //Allocate data on unified memory so it is accessible from CPU or GPU
//   gpuErrchk(cudaMallocManaged(&cls, sizeof(_PDM_closest_t)));
//   gpuErrchk(cudaMallocManaged(&octree_id, sizeof(int)));
//   printf("After malloc\n");

//   cls = _get_from_id (id);

//   printf("after cls\n");

//   double b_t_elapsed;
//   double b_t_cpu;
//   double b_t_cpu_u;
//   double b_t_cpu_s;

//   cls->times_elapsed[BEGIN] = PDM_timer_elapsed(cls->timer);
//   cls->times_cpu[BEGIN]     = PDM_timer_cpu(cls->timer);
//   cls->times_cpu_u[BEGIN]   = PDM_timer_cpu_user(cls->timer);
//   cls->times_cpu_s[BEGIN]   = PDM_timer_cpu_sys(cls->timer);

//   b_t_elapsed = cls->times_elapsed[BEGIN];
//   b_t_cpu     = cls->times_cpu[BEGIN];
//   b_t_cpu_u   = cls->times_cpu_u[BEGIN];
//   b_t_cpu_s   = cls->times_cpu_s[BEGIN];
//   PDM_timer_resume(cls->timer);


//   int i_rank;
//   PDM_MPI_Comm_rank (cls->comm, &i_rank);

//   //-->GPU
//   const int depth_max = 31;//?
//   const int points_in_leaf_max = 1;//2*cls->n_closest;//?
//   const int build_leaf_neighbours = 1;

//   printf("Before create\n");
//   /* Create empty parallel octree structure */
//   *octree_id = PDM_para_octree_create_GPU (cls->src_cloud->n_part,
//                                           depth_max,
//                                           points_in_leaf_max,
//                                           build_leaf_neighbours,
//                                           cls->comm);

//   printf("gpu print:\n");
//   print_from_gpu<<<1,1>>>(*octree_id);
//   cudaDeviceSynchronize();
//   printf("after gpu print\n");

//   /* Set source point clouds */
//   for (int i_part = 0; i_part < cls->src_cloud->n_part; i_part++) {
//     PDM_para_octree_point_cloud_set (*octree_id,
//                                      i_part,
//                                      cls->src_cloud->n_points[i_part],
//                                      cls->src_cloud->coords[i_part],
//                                      cls->src_cloud->gnum[i_part]);
//   }


//   /* Build parallel octree */
//   PDM_para_octree_build (*octree_id);
//   //PDM_para_octree_dump (octree_id);
//   PDM_para_octree_dump_times (*octree_id);
//   //<--


//   // /* Concatenate partitions */
//   // int n_tgt = 0;
//   // for (int i_part = 0; i_part < cls->tgt_cloud->n_part; i_part++)
//   //   n_tgt += cls->tgt_cloud->n_points[i_part];

//   // double      *tgt_coord = malloc (sizeof(double)      * n_tgt * 3);
//   // PDM_g_num_t *tgt_g_num = malloc (sizeof(PDM_g_num_t) * n_tgt);
//   // PDM_g_num_t *closest_src_gnum = malloc (sizeof(PDM_g_num_t) * n_tgt * cls->n_closest);
//   // double      *closest_src_dist = malloc (sizeof(double)      * n_tgt * cls->n_closest);

//   // n_tgt = 0;
//   // for (int i_part = 0; i_part < cls->tgt_cloud->n_part; i_part++) {
//   //   for (int i = 0; i < cls->tgt_cloud->n_points[i_part]; i++) {
//   //     for (int j = 0; j < 3; j++)
//   //       tgt_coord[n_tgt + 3*i + j] = cls->tgt_cloud->coords[i_part][3*i + j];
//   //     tgt_g_num[n_tgt + i] = cls->tgt_cloud->gnum[i_part][i];
//   //   }
//   //   n_tgt += cls->tgt_cloud->n_points[i_part];
//   // }
  

//   // /* Search closest source points from target points */
//   // PDM_para_octree_closest_point (octree_id,
//   //                                cls->n_closest,
//   //                                n_tgt,
//   //                                tgt_coord,
//   //                                tgt_g_num,
//   //                                closest_src_gnum,
//   //                                closest_src_dist);


//   // /* Restore partitions */
//   // free (tgt_coord);
//   // free (tgt_g_num);
//   // n_tgt = 0;

//   // cls->tgt_cloud->closest_src_gnum = malloc (sizeof(PDM_g_num_t *) * cls->tgt_cloud->n_part);
//   // cls->tgt_cloud->closest_src_dist = malloc (sizeof(double *)      * cls->tgt_cloud->n_part);

//   // for (int i_part = 0; i_part < cls->tgt_cloud->n_part; i_part++) {
//   //   int s_closest_src = cls->n_closest * cls->tgt_cloud->n_points[i_part];

//   //   cls->tgt_cloud->closest_src_gnum[i_part] = malloc (sizeof(PDM_g_num_t) * s_closest_src);
//   //   cls->tgt_cloud->closest_src_dist[i_part] = malloc (sizeof(double)      * s_closest_src);

//   //   for (int i = 0; i < cls->tgt_cloud->n_points[i_part]; i++) {
//   //     for (int j = 0; j < cls->n_closest; j++) {
//   //       cls->tgt_cloud->closest_src_gnum[i_part][cls->n_closest*i+j] =
//   //         closest_src_gnum[n_tgt + cls->n_closest*i + j];

//   //       cls->tgt_cloud->closest_src_dist[i_part][cls->n_closest*i+j] =
//   //         closest_src_dist[n_tgt + cls->n_closest*i + j];
//   //     }
//   //   }
//   //   n_tgt += cls->n_closest * cls->tgt_cloud->n_points[i_part];
//   // }
//   // free (closest_src_gnum);
//   // free (closest_src_dist);



//   //-->GPU
//   /* Free parallel octree */
//   PDM_para_octree_free_GPU (*octree_id);
//   gpuErrchk(cudaFree(octree_id));
//   //<--


//   PDM_timer_hang_on(cls->timer);

//   cls->times_elapsed[END] = PDM_timer_elapsed(cls->timer);
//   cls->times_cpu[END]     = PDM_timer_cpu(cls->timer);
//   cls->times_cpu_u[END]   = PDM_timer_cpu_user(cls->timer);
//   cls->times_cpu_s[END]   = PDM_timer_cpu_sys(cls->timer);

//   b_t_elapsed = cls->times_elapsed[END];
//   b_t_cpu     = cls->times_cpu[END];
//   b_t_cpu_u   = cls->times_cpu_u[END];
//   b_t_cpu_s   = cls->times_cpu_s[END];
//   PDM_timer_resume(cls->timer);
// }


// #ifdef	__cplusplus
// }
// #endif
// #undef NTIMER
